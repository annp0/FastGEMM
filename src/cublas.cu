#pragma once
#include <hip/hip_runtime.h>


#include <hipblas.h>

void run_cublas(hipblasHandle_t handle, int M, int N, int K, float alpha, float beta, float *A, float *B, float *C)
{
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F, N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}