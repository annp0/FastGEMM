#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>

#include "cuda_check.cuh"

const int WARPSIZE = 32;

template <const int BM, const int BN, const int BK, const int rowStrideA,
          const int rowStrideB>
__device__ void loadFromGmem(int N, int K, const float *A, const float *B,
                             float *As, float *Bs, int innerRowA, int innerColA,
                             int innerRowB, int innerColB) {
  for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
    const float4 tmp = reinterpret_cast<const float4 *>(
        &A[(innerRowA + offset) * K + innerColA * 4])[0];
    As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
    As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
    As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
    As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
  }

  for (uint offset = 0; offset + rowStrideB <= BK; offset += rowStrideB) {
    reinterpret_cast<float4 *>(
        &Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
        reinterpret_cast<const float4 *>(
            &B[(innerRowB + offset) * N + innerColB * 4])[0];
  }
}

template <const int BM, const int BN, const int BK, const int WM, const int WN,
          const int WMITER, const int WNITER, const int WSUBM, const int WSUBN,
          const int TM, const int TN>
__device__ void
processFromSmem(float *regM, float *regN, float *threadResults, const float *As,
                const float *Bs, const uint warpRow, const uint warpCol,
                const uint threadRowInWarp, const uint threadColInWarp) {
  for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
    for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
      for (uint i = 0; i < TM; ++i) {
        regM[wSubRowIdx * TM + i] = 
            As[(dotIdx * BM) + warpRow * WM + wSubRowIdx * WSUBM + threadRowInWarp * TM + i];
      }
    }
    for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
      for (uint i = 0; i < TN; ++i) {
        regN[wSubColIdx * TN + i] =
            Bs[(dotIdx * BN) + warpCol * WN + wSubColIdx * WSUBN + threadColInWarp * TN + i];
      }
    }

    for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
      for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
          for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
            threadResults[(wSubRowIdx * TM + resIdxM) * (WNITER * TN) + (wSubColIdx * TN) + resIdxN] 
                += regM[wSubRowIdx * TM + resIdxM] * regN[wSubColIdx * TN + resIdxN];
          }
        }
      }
    }
  }
}


template <const int BM, const int BN, const int BK, const int WM, const int WN,
          const int WNITER, const int TM, const int TN, const int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
    sgemm_optimized(int M, int N, int K, float alpha, float *A, float *B,
                    float beta, float *C) {
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;

  const uint warpIdx = threadIdx.x / WARPSIZE;
  const uint warpCol = warpIdx % (BN / WN);
  const uint warpRow = warpIdx / (BN / WN);

  constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
  constexpr uint WSUBM = WM / WMITER; // 64/2=32
  constexpr uint WSUBN = WN / WNITER; // 32/2=16

  const uint threadIdxInWarp = threadIdx.x % WARPSIZE;
  const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN);
  const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN);

  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];

  A += cRow * BM * K;
  B += cCol * BN;
  C += (cRow * BM + warpRow * WM) * N + cCol * BN + warpCol * WN;

  const uint innerRowA = threadIdx.x / (BK / 4);
  const uint innerColA = threadIdx.x % (BK / 4);
  constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;
  const uint innerRowB = threadIdx.x / (BN / 4);
  const uint innerColB = threadIdx.x % (BN / 4);
  constexpr uint rowStrideB = NUM_THREADS / (BN / 4);

  float threadResults[WMITER * TM * WNITER * TN] = {0.0};
  float regM[WMITER * TM] = {0.0};
  float regN[WNITER * TN] = {0.0};

  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    loadFromGmem<BM, BN, BK, rowStrideA, rowStrideB>(N, K, A, B, As, Bs, innerRowA, innerColA, innerRowB, innerColB);
    __syncthreads();
    processFromSmem<BM, BN, BK, WM, WN, WMITER, WNITER, WSUBM, WSUBN, TM, TN>
        (regM, regN, threadResults, As, Bs, warpRow, warpCol, threadRowInWarp, threadColInWarp);
    A += BK; 
    B += BK * N;
    __syncthreads();
  }

  for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
    for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
      float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
      for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
        for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
          float4 tmp = reinterpret_cast<float4 *>(
              &C_interim[(threadRowInWarp * TM + resIdxM) * N + threadColInWarp * TN + resIdxN])[0];
          const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) + wSubColIdx * TN + resIdxN;
          tmp.x = alpha * threadResults[i + 0] + beta * tmp.x;
          tmp.y = alpha * threadResults[i + 1] + beta * tmp.y;
          tmp.z = alpha * threadResults[i + 2] + beta * tmp.z;
          tmp.w = alpha * threadResults[i + 3] + beta * tmp.w;
          reinterpret_cast<float4 *>(&C_interim[(threadRowInWarp * TM + resIdxM) * N + threadColInWarp * TN + resIdxN])[0] = tmp;
        }
      }
    }
  }
}

void run_sgemm_optimized(int M, int N, int K, float alpha, float beta, float *A, float *B, float *C) {
  const uint NUM_THREADS = 128;
  const uint BN = 128;
  const uint BM = 128;
  const uint BK = 16;
  const uint WN = 64;
  const uint WM = 64;
  const uint WNITER = 4;
  const uint TN = 4;
  const uint TM = 8;
  dim3 blockDim(NUM_THREADS);
  dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
  sgemm_optimized<BM, BN, BK, WM, WN, WNITER, TM, TN, NUM_THREADS><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}